
#include <hip/hip_runtime.h>
#include <assert.h>
#include <errno.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef signed char schar;
typedef unsigned char uchar;
typedef short shrt;
typedef unsigned short ushrt;
typedef unsigned uint;
typedef unsigned long ulong;
typedef long long llong;
typedef unsigned long long ullong;

typedef float flt;
typedef double dbl;
typedef long double ldbl;

#define exit_if(cnd_value, msg) \
    do { \
        if (cnd_value) \
        { \
            if (errno) \
                perror(msg); \
            else \
                fprintf(stderr, "error: %s\n", msg); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define cudaErrorCheck(error) \
    do { \
        hipError_t res = error; \
        if (res != hipSuccess) \
        { \
            fprintf(stderr, "cuda %s:%d error: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(res)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define NUM_BLOCKS (1024)
#define BLOCK_SIZE (1024)

__global__ void kernel(dbl * const __restrict__ first,
    const dbl * const __restrict__ second,
    const size_t n)
{
    const size_t offset = gridDim.x * blockDim.x;
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < n)
    {
        first[idx] -= second[idx];
        idx += offset;
    }
}

int main(void)
{
    size_t n;
    scanf("%zu", &n);
    dbl * const first  = (dbl *) malloc(sizeof(dbl) * n),
        * const second = (dbl *) malloc(sizeof(dbl) * n);
    exit_if(!first || !second, "malloc()");
    memset(first, 0, n * sizeof(dbl));
    memset(second, 0, n * sizeof(dbl));

    dbl *device_first, *device_second;
    cudaErrorCheck(hipMalloc(&device_first, sizeof(dbl) * n));
    cudaErrorCheck(hipMemcpy(device_first, first, sizeof(dbl) * n,
        hipMemcpyHostToDevice));
    cudaErrorCheck(hipMalloc(&device_second, sizeof(dbl) * n));
    cudaErrorCheck(hipMemcpy(device_second, second, sizeof(dbl) * n,
        hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    cudaErrorCheck(hipEventCreate(&start));
    cudaErrorCheck(hipEventCreate(&stop));
    cudaErrorCheck(hipEventRecord(start, 0));

    kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(device_first, device_second, n);
    cudaErrorCheck(hipGetLastError());

    cudaErrorCheck(hipEventRecord(stop, 0));
    cudaErrorCheck(hipEventSynchronize(stop));

    flt time;
    cudaErrorCheck(hipEventElapsedTime(&time, start, stop));
    cudaErrorCheck(hipEventDestroy(start));
    cudaErrorCheck(hipEventDestroy(stop));
    printf("time = %f\n", time);

    cudaErrorCheck(hipMemcpy(first, device_first, sizeof(dbl) * n,
        hipMemcpyDeviceToHost));
    cudaErrorCheck(hipFree(device_first));
    cudaErrorCheck(hipFree(device_second));

    free(first);
    free(second);

    return 0;
}
